#include "hip/hip_runtime.h"
#include "compression.h"

#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cudaCompress/Instance.h>
#include <cudaCompress/Encode.h>
#include <cudaCompress/util/Bits.h>
#include <cudaCompress/util/DWT.h>
#include <cudaCompress/util/Quantize.h>
#include <cudaCompress/util/YCoCg.h>
#include <cudaCompress/Timing.h>
using namespace cudaCompress;


#include "../cudaCompress/src/examples/tthread/tinythread.h"

#include "../cudaCompress/src/examples/tools/entropy.h"
#include "../cudaCompress/src/examples/tools/imgtools.h"
#include "../cudaCompress/src/examples/tools/rawfile.h"

#include "../cudaCompress/src/examples/cudaUtil.h"

#include "../cudaCompress/src/examples/CompressImage.h"
#include "../cudaCompress/src/examples/CompressHeightfield.h"
#include "../cudaCompress/src/examples/CompressVolume.h"

void compress(int4 size, float * h_data, std::vector<uint> & bitStreams, const float & Quant_step)
{
	// a device pointer to the data sitted on GPU
	float* d_data = nullptr;

	const bool doRLEOnlyOnLvl0 = true;

	const unsigned int elemCountTotal = size.x*size.y*size.z*size.w;
	

	// Allocate GPU Memory
	hipMalloc(&d_data, elemCountTotal * sizeof(float));
	
	// Copy the uncompressed data into memory
	hipMemcpy(d_data, h_data, elemCountTotal * sizeof(float), hipMemcpyHostToDevice);


	uint huffmanBits = 0;


	GPUResources::Config config = CompressVolumeResources::getRequiredResources(size.x, size.y, size.z, 1, huffmanBits);
	GPUResources shared;

	shared.create(config);
	CompressVolumeResources res;
	res.create(shared.getConfig());

	// To keep the compressed 
	

	
	compressVolumeFloat(shared, res, d_data, size.x, size.y, size.z, 2, bitStreams, Quant_step, doRLEOnlyOnLvl0);
	


	//// Set the device memory to 0
	//cudaSafeCall(hipMemset(d_data, 0, elemCountTotal * sizeof(float)));

	//// Pinning the Host memory
	//cudaSafeCall(hipHostRegister(bitStreams.data(), bitStreams.size() * sizeof(uint), hipHostRegisterDefault));
	//




	//decompressVolumeFloat(shared, res, d_data, size.x, size.y, size.z, 2, bitStreams, 0.01f, doRLEOnlyOnLvl0);



	hipFree(d_data);

	res.destroy();
	shared.destroy();
	 
}


